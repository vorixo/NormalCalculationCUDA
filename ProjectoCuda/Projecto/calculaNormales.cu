#include "hip/hip_runtime.h"
/*----------------------------------------------------------------------------*/
/*  FICHERO:       calculaNormales.cu									        */
/*  AUTOR:         Jorge Azorin													*/
/*																				*/
/*	IMPLEMENTACION GPU:															*/
/*		Jordi Amor�s															*/
/*		Alvaro Jover															*/
/*		Alejandro																*/
/*		Hector																	*/
/*																				*/
/*  RESUMEN																		*/
/*  ~~~~~~~																		*/
/* Ejercicio grupal para el c�lculo de las normales de una superficie			*/
/*----------------------------------------------------------------------------*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>


// includes, project
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "calculaNormales.h"
#include <Windows.h>


typedef LARGE_INTEGER timeStamp;
double getTime();

double noTransfer = 0;

/*----------------------------------------------------------------------------*/
/*  FUNCION A PARALELIZAR  (versi�n secuencial-CPU)  				          */
/*	C�lculo de las normales de una superficie definida por una                */
/*  una malla de vtotal x utotal puntos 3D                                    */
/*----------------------------------------------------------------------------*/
int CalculoNormalesCPU()
{
	TPoint3D direct1, direct2, normal;
	int vecindadU[9] = { -1, 0, 1, 1, 1, 0, -1, -1, -1 }; // Vecindad 8 + 1 para calcular todas las rectas
	int vecindadV[9] = { -1, -1, -1, 0, 1, 1, 1, 0, -1 };
	int vV, vU;
	int numDir;
	int oKdir1, oKdir2;
	/* La vencidad es:
	*--*--*
	|  |  |
	*--X--*
	|  |  |
	*--*--*
	*/
	int cont = 0;

	for (int u = 0; u<S.UPoints; u++)			// Recorrido de todos los puntos de la superficie
	{
		for (int v = 0; v<S.VPoints; v++)
		{
			normal.x = 0;
			normal.y = 0;
			normal.z = 0;
			numDir = 0;
			for (int nv = 0; nv < 8; nv++)  // Para los puntos de la vecindad
			{
				vV = v + vecindadV[nv];
				vU = u + vecindadU[nv];
				if (vV >= 0 && vU >= 0 && vV<S.VPoints && vU<S.UPoints)
				{
					direct1.x = S.Buffer[v][u].x - S.Buffer[vV][vU].x;
					direct1.y = S.Buffer[v][u].y - S.Buffer[vV][vU].y;
					direct1.z = S.Buffer[v][u].z - S.Buffer[vV][vU].z;
					oKdir1 = 1;
				}
				else
				{
					direct1.x = 0.0;
					direct1.y = 0.0;
					direct1.z = 0.0;
					oKdir1 = 0;
				}
				vV = v + vecindadV[nv + 1];
				vU = v + vecindadU[nv + 1];

				if (vV >= 0 && vU >= 0 && vV<S.VPoints && vU<S.UPoints)
				{
					direct2.x = S.Buffer[v][u].x - S.Buffer[vV][vU].x;
					direct2.y = S.Buffer[v][u].y - S.Buffer[vV][vU].y;
					direct2.z = S.Buffer[v][u].z - S.Buffer[vV][vU].z;
					oKdir2 = 1;
				}
				else
				{
					direct2.x = 0.0;
					direct2.y = 0.0;
					direct2.z = 0.0;
					oKdir2 = 0;
				}
				if (oKdir1 == 1 && oKdir2 == 1)
				{
					normal.x += direct1.y*direct2.z - direct1.z*direct2.y;
					normal.y += direct1.x*direct2.z - direct1.z*direct2.x;
					normal.z += direct1.x*direct2.y - direct1.y*direct2.x;
					numDir++;
				}
			}
			NormalUCPU[cont] = normal.x / (float)numDir;
			NormalVCPU[cont] = normal.y / (float)numDir;
			NormalWCPU[cont] = normal.z / (float)numDir;
			cont++;
		}
	}

	return OKCALC;									// Simulaci�n CORRECTA
}



// ---------------------------------------------------------------
// ---------------------------------------------------------------
// FUNCION A IMPLEMENTAR POR EL GRUPO (paralelizaci�n de CalculoNormalesCPU)
// ---------------------------------------------------------------
// ---------------------------------------------------------------


__global__ void getNormal(TPoint3D *d_Buffer, float *d_NormalUGPU, float *d_NormalVGPU, float *d_NormalWGPU, int U, int V) {

	int id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < U*V) {

		int d_vU[9] = { -1, 0, 1, 1, 1, 0, -1, -1, -1 };
		int d_vV[9] = { -1, -1, -1, 0, 1, 1, 1, 0, -1 };

		int vecindad, oKdir1, oKdir2, numDir = 0, v, u, vV, vU;

		TPoint3D normal, direct1, direct2;
		normal.x = 0;
		normal.y = 0;
		normal.z = 0;

		for (unsigned nv = 0; nv < 8; nv++) {

			v = id % V;		//get row
			u = id / V;		//get column

			vV = v + d_vV[nv];
			vU = u + d_vU[nv];

			if (vV >= 0 && vU >= 0 && vV<V && vU<U) {
				vecindad = vU * V + vV;

				direct1.x = d_Buffer[id].x - d_Buffer[vecindad].x;
				direct1.y = d_Buffer[id].y - d_Buffer[vecindad].y;
				direct1.z = d_Buffer[id].z - d_Buffer[vecindad].z;
				oKdir1 = 1;
			}
			else
			{
				direct1.x = 0.0;
				direct1.y = 0.0;
				direct1.z = 0.0;
				oKdir1 = 0;
			}

			vV = v + d_vV[nv + 1];
			vU = v + d_vU[nv + 1];

			if (vV >= 0 && vU >= 0 && vV<V && vU<U) {
				vecindad = vU * V + vV;
				direct2.x = d_Buffer[id].x - d_Buffer[vecindad].x;
				direct2.y = d_Buffer[id].y - d_Buffer[vecindad].y;
				direct2.z = d_Buffer[id].z - d_Buffer[vecindad].z;
				oKdir2 = 1;
			}
			else
			{
				direct2.x = 0.0;
				direct2.y = 0.0;
				direct2.z = 0.0;
				oKdir2 = 0;
			}

			if (oKdir1 == 1 && oKdir2 == 1) {
				normal.x += direct1.y * direct2.z - direct1.z * direct2.y;
				normal.y += direct1.x * direct2.z - direct1.z * direct2.x;
				normal.z += direct1.x * direct2.y - direct1.y * direct2.x;
				numDir++;
			}
		}

		d_NormalUGPU[id] = normal.x / (float)numDir;
		d_NormalVGPU[id] = normal.y / (float)numDir;
		d_NormalWGPU[id] = normal.z / (float)numDir;
	}
}

int CalculoNormalesGPU()
{
	unsigned U = S.UPoints;
	unsigned V = S.VPoints;

	double time, end_time;
	//Problema para computar el algoritmo teniendo la malla 3D aplanada en un vector de 1 Dimension

	/* ------------> S.UPoints (u)
	| 0	 3  6  9
	| 1	 4  7  10
	| 2	 5  8  11
	v
	S.VPoints (v)

	Esto pasado a h_Buffer (unidimensional) queda:

	------------------------> S.UPoints * S.Vpoints (id)
	0 1 2 3 4 5 6 7 8 9 10 11

	Obtendremos los dos indices 'v' y 'u' a partir del indice unidimensional:

	v = id % S.VPoints
	u = id / S.VPoints

	*/

	//Allocated in CPU: S.Buffer (**TPoint3D) -> Flattened to -> h_Buffer (*TPoint3D)

	TPoint3D *h_Buffer;
	h_Buffer = (TPoint3D *)malloc(sizeof(TPoint3D)*U*V);

	//Flattening **S.Buffer to *h_Buffer
	unsigned k = 0;
	for (unsigned i = 0; i< U; i++) {
		for (unsigned j = 0; j < V; j++) {
			h_Buffer[k] = S.Buffer[j][i];
			k++;
		}
	}

	//Allocated in GPU: d_Buffer;
	TPoint3D *d_Buffer;

	/*
	Allocated in CPU:
	NormalVGPU
	NormalUGPU
	NormalWGPU

	Allocated in GPU:
	d_NormalVGPU
	d_NormalUGPU
	d_NormalWGPU
	*/

	float *d_NormalVGPU;
	float *d_NormalUGPU;
	float *d_NormalWGPU;

	//Allocate on device memory for 3D Surface and the 3 normal vectors (result)
	hipMalloc(&d_Buffer, U*V * sizeof(TPoint3D));
	hipMalloc(&d_NormalVGPU, sizeof(float)*U*V);
	hipMalloc(&d_NormalUGPU, sizeof(float)*U*V);
	hipMalloc(&d_NormalWGPU, sizeof(float)*U*V);

	//Copy to device 3D Surface
	hipMemcpy(d_Buffer, h_Buffer, sizeof(TPoint3D)* U*V, hipMemcpyHostToDevice);

	//C�lculo del tiempo de ejecuci�n del algoritmo sin tener en cuenta data transfer
	time = getTime();

	getNormal << < U*V / 512 + 1, 512 >> >(d_Buffer, d_NormalUGPU, d_NormalVGPU, d_NormalWGPU, U, V);

	end_time = getTime();
	noTransfer = (end_time - time);

	hipMemcpy(NormalVGPU, d_NormalVGPU, U*V * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(NormalUGPU, d_NormalUGPU, U*V * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(NormalWGPU, d_NormalWGPU, U*V * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_Buffer);
	hipFree(d_NormalVGPU);
	hipFree(d_NormalUGPU);
	hipFree(d_NormalWGPU);

	return OKCALC;
}

// Declaraciones adelantadas de funciones
int LeerSuperficie(const char *fichero);



////////////////////////////////////////////////////////////////////////////////
//PROGRAMA PRINCIPAL
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char** argv)
{

	double gpu_start_time, gpu_end_time;
	double cpu_start_time, cpu_end_time;

	/* Numero de argumentos */
	if (argc != 2)
	{
		fprintf(stderr, "Numero de parametros incorecto\n");
		fprintf(stderr, "Uso: %s superficie\n", argv[0]);
		return;
	}

	/* Apertura de Fichero */
	printf("C�lculo de las normales de la superficie...\n");
	/* Datos de la superficie */
	if (LeerSuperficie((char *)argv[1]) == ERRORCALC)
	{
		fprintf(stderr, "Lectura de superficie incorrecta\n");
		return;
	}
	int numPuntos;
	numPuntos = S.UPoints*S.VPoints;

	printf(" Alto: %i\n Ancho: %i\n", S.VPoints, S.UPoints);

	// Creaci�n buffer resultados para versiones CPU y GPU
	NormalVCPU = (float*)malloc(numPuntos*sizeof(float));
	NormalUCPU = (float*)malloc(numPuntos*sizeof(float));
	NormalWCPU = (float*)malloc(numPuntos*sizeof(float));
	NormalVGPU = (float*)malloc(numPuntos*sizeof(float));
	NormalUGPU = (float*)malloc(numPuntos*sizeof(float));
	NormalWGPU = (float*)malloc(numPuntos*sizeof(float));

	/* Algoritmo a paralelizar */
	cpu_start_time = getTime();
	if (CalculoNormalesCPU() == ERRORCALC)
	{
		fprintf(stderr, "C�lculo CPU incorrecta\n");
		BorrarSuperficie();
		if (NormalVCPU != NULL) free(NormalVCPU);
		if (NormalUCPU != NULL) free(NormalUCPU);
		if (NormalWCPU != NULL) free(NormalUCPU);
		if (NormalVGPU != NULL) free(NormalVGPU);
		if (NormalWGPU != NULL) free(NormalVGPU);
		if (NormalUGPU != NULL) free(NormalUGPU);		exit(1);
	}
	cpu_end_time = getTime();
	/* Algoritmo a implementar */
	gpu_start_time = getTime();
	if (CalculoNormalesGPU() == ERRORCALC)
	{
		fprintf(stderr, "C�lculo GPU incorrecta\n");
		BorrarSuperficie();
		if (NormalVCPU != NULL) free(NormalVCPU);
		if (NormalUCPU != NULL) free(NormalUCPU);
		if (NormalWCPU != NULL) free(NormalUCPU);
		if (NormalVGPU != NULL) free(NormalVGPU);
		if (NormalUGPU != NULL) free(NormalUGPU);
		if (NormalVGPU != NULL) free(NormalVGPU);
		return;
	}
	gpu_end_time = getTime();
	// Comparaci�n de correcci�n
	int comprobar = OKCALC;

	int fallos = 0;
	for (int i = 0; i<numPuntos; i++)
	{
		if (((int)NormalVCPU[i] * 1000 != (int)NormalVGPU[i] * 1000) || ((int)NormalUCPU[i] * 1000 != (int)NormalUGPU[i] * 1000) || ((int)NormalWCPU[i] * 1000 != (int)NormalWGPU[i] * 1000))
		{
			comprobar = ERRORCALC;
			fprintf(stderr, "Fallo en el punto %d, valor correcto V=%f U=%f W=%f\n", i, NormalVCPU[i], NormalUCPU[i], NormalWCPU[i]);
			printf("Fallo en el punto %d, valor obtenido V=%f U=%f W=%f \n", i, NormalVGPU[i], NormalUGPU[i], NormalWGPU[i]);
			fallos++;
		}
	}

	// Impresion de resultados
	if (comprobar == OKCALC)
	{
		printf("C�lculo correcto!\n");

	}
	// Impresi�n de resultados
	printf("Tiempo ejecuci�n GPU : %fs\n", \
		gpu_end_time - gpu_start_time);
	printf("Tiempo de ejecuci�n en la CPU : %fs\n", \
		cpu_end_time - cpu_start_time);
	printf("Se ha conseguido un factor de aceleraci�n %fx utilizando CUDA\n", (cpu_end_time - cpu_start_time) / (gpu_end_time - gpu_start_time));
	printf("Se ha conseguido un factor de aceleraci�n %fx utilizando CUDA (sin tener en cuenta transferencia de datos)\n", (cpu_end_time - cpu_start_time) / noTransfer);
	// Limpieza de buffers
	BorrarSuperficie();
	if (NormalVCPU != NULL) free(NormalVCPU);
	if (NormalUCPU != NULL) free(NormalUCPU);
	if (NormalWCPU != NULL) free(NormalWCPU);
	if (NormalVGPU != NULL) free(NormalVGPU);
	if (NormalUGPU != NULL) free(NormalUGPU);
	if (NormalWGPU != NULL) free(NormalWGPU);
	return;
}

int
main(int argc, char** argv)
{
	runTest(argc, argv);

	int device;
	hipGetDevice(&device);

	struct hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, device);

	printf("\nDevice properties: \n");

	printf("  Device name: %s\n", prop.name);
	printf("  Memory Clock Rate (KHz): %d\n",
		prop.memoryClockRate);
	printf("  Memory Bus Width (bits): %d\n",
		prop.memoryBusWidth);
	printf("  Peak Memory Bandwidth (GB/s): %f\n",
		2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);
	printf("  Clock Rate (KHz): %i\n", prop.clockRate);
	printf("  Total Global Memory (MB): %i\n\n", prop.totalGlobalMem / 1048576);

	getchar();
}

/* Funciones auxiliares */
double getTime()
{
	timeStamp start;
	timeStamp dwFreq;
	QueryPerformanceFrequency(&dwFreq);
	QueryPerformanceCounter(&start);
	return double(start.QuadPart) / double(dwFreq.QuadPart);
}



/*----------------------------------------------------------------------------*/
/*	Funci�n:  LeerSuperficie(char *fichero)						              */
/*													                          */
/*	          Lee los datos de la superficie de un fichero con formato .FOR   */
/*----------------------------------------------------------------------------*/
int LeerSuperficie(const char *fichero)
{
	int i, j, count;		/* Variables de bucle */
	int utotal, vtotal;		/* Variables de tama�o de superficie */
	FILE *fpin; 			/* Fichero */
	double x, y, z;

	/* Apertura de Fichero */
	if ((fpin = fopen(fichero, "r")) == NULL) return ERRORCALC;
	/* Lectura de cabecera */
	if (fscanf(fpin, "Ancho=%d\n", &utotal)<0) return ERRORCALC;
	if (fscanf(fpin, "Alto=%d\n", &vtotal)<0) return ERRORCALC;
	if (utotal*vtotal <= 0) return ERRORCALC;
	/* Localizacion de comienzo */
	if (feof(fpin)) return ERRORCALC;
	/* Inicializaci�n de parametros geometricos */
	if (CrearSuperficie(utotal, vtotal) == ERRORCALC) return ERRORCALC;
	/* Lectura de coordenadas */
	count = 0;
	for (i = 0; i<utotal; i++)
	{
		for (j = 0; j<vtotal; j++)
		{
			if (!feof(fpin))
			{
				fscanf(fpin, "%lf %lf %lf\n", &x, &y, &z);
				S.Buffer[j][i].x = x;
				S.Buffer[j][i].y = y;
				S.Buffer[j][i].z = z;
				count++;
			}
			else break;
		}
	}
	fclose(fpin);
	if (count != utotal*vtotal) return ERRORCALC;

	return OKCALC;
}
